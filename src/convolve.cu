#include "hip/hip_runtime.h"
#include <config.h>
#include <cassert>

/* nvcc compiles *.cu files as C++, whereas the remainder of our source code
 * is in C; this ensures that we get C external linkage so that we can easily
 * call the functions in this file from C code, not just C++ code */
extern "C" {

__global__ void kernel_convolve_v0(const int m, const int n, const float* p,
    const int incp, const float* q, const int incq, float* r,
    const int incr) {
  /* element of r for which thread is responsible */
  int i = threadIdx.x + blockIdx.x*blockDim.x;

  if (i < m + n - 1) {
    float result = 0.0f;
    for (int j = 0; j < n; ++j) {
      if (0 <= i - j && i - j < m) {
        result += p[(i - j)*incp]*q[j*incq];
      }
    }
    r[i] = result;
  }
}

__global__ void kernel_convolve_v1(const int m, const int n, const float* p,
    const int incp, const float* q, const int incq, float* r,
    const int incr) {
  /* element of r for which thread is responsible */
  int i = threadIdx.x + blockIdx.x*blockDim.x;
  
  if (i < m) {
    float result1 = 0.0f, result2 = 0.0f;
    for (int j = 0; j < n; ++j) {
      if (0 <= i - j) {
        result1 += p[(i - j)*incp]*q[j*incq];
      } else {
        result2 += p[(m + i - j)*incp]*q[j*incq];
      }
    }
    r[i] = result1;
    if (i < n - 1) {
      r[i + m] = result2;
    }
  }
}

__global__ void kernel_convolve_v2(const int m, const int n, const float* p,
    const int incp, const float* q, const int incq, float* r,
    const int incr) {
  extern __shared__ float q_shared[];

  /* element of r for which thread is responsible */
  int i = threadIdx.x + blockIdx.x*blockDim.x;

  float result1 = 0.0f, result2 = 0.0f;
  for (int base_j = 0; base_j < n; base_j += blockDim.x) {
    int j = threadIdx.x;
    __syncthreads();
    q_shared[j] = (base_j + j < n) ? q[(base_j + j)*incq] : 0.0f;
    __syncthreads();

    for (j = 0; j < blockDim.x; ++j) {
      if (0 <= i - base_j - j) {
        result1 += p[(i - base_j - j)*incp]*q_shared[j];
      } else {
        result2 += p[(m + i - base_j - j)*incp]*q_shared[j];
      }
    }
  }
  if (i < m) {
    r[i] = result1;
    if (i < n - 1) {
      r[i + m] = result2;
    }
  }
}

__global__ void kernel_convolve_v3(const int m, const int n, const float* p,
    const int incp, const float* q, const int incq, float* r,
    const int incr) {
  extern __shared__ float shared[];
  float* q_shared = shared;
  float* p_shared = q_shared + 2*blockDim.x;  // permits -ve indices

  int i = threadIdx.x;  
  int j = threadIdx.x;
  float result1 = 0.0f, result2 = 0.0f;
  for (int base_i = blockIdx.x*blockDim.x, base_j = 0; base_j < n;
      base_i -= blockDim.x, base_j += blockDim.x) {
    __syncthreads();
    q_shared[j] = (base_j + j < n) ? q[(base_j + j)*incq] : 0.0f;
    p_shared[i] = p[((base_i + i + m) % m)*incp];
    p_shared[i - blockDim.x] = p[((base_i + i - blockDim.x + m) % m)*incp];
    __syncthreads();

    for (int k = 0; k < blockDim.x; ++k) {
      if (0 <= base_i + i - k) {
        result1 += p_shared[i - k]*q_shared[k];
      } else {
        result2 += p_shared[i - k]*q_shared[k];        
      }
    }
  }

  /* element of r for which thread is responsible */
  i = threadIdx.x + blockIdx.x*blockDim.x;
  if (i < m) {
    r[i] = result1;
    if (i < n - 1) {
      r[i + m] = result2;
    }
  }
}

__global__ void kernel_convolve_v4(const int m, const int n, const float* p,
    const int incp, const float* q, const int incq, float* r,
    const int incr) {
  assert(blockDim.x == warpSize && gridDim.x == 1);

  extern __shared__ float shared[];
  float* q_shared = shared;
  float* p_shared = q_shared + 2*warpSize*blockDim.y;  // permits -ve indices

  int i = threadIdx.y*warpSize + threadIdx.x;
  int j = threadIdx.y*warpSize + threadIdx.x;
  int l = threadIdx.y;
  float result1 = 0.0f, result2 = 0.0f;
  for (int base_i = blockIdx.y*blockDim.y, base_j = 0; base_j < n;
      base_i -= warpSize*blockDim.y, base_j += warpSize*blockDim.y) {
    __syncthreads();
    q_shared[j] = (base_j + j < n) ? q[(base_j + j)*incq] : 0.0f;
    p_shared[i] = p[((base_i + i + m) % m)*incp];
    p_shared[i - warpSize*blockDim.y] = p[((base_i + i - warpSize*blockDim.y + m) % m)*incp];
    __syncthreads();

    for (int k = threadIdx.x; k < warpSize*blockDim.y; k += warpSize) {
      if (0 <= base_i + l - k) {
        result1 += p_shared[l - k]*q_shared[k];
      } else {
        result2 += p_shared[l - k]*q_shared[k];        
      }
    }
  }

  /* sum across threads of warp, using butterfly sum */
  for (int k = 16; k >= 1; k /= 2) {
    result1 += __shfl_xor_sync(0xffffffff, result1, k, warpSize);
    result2 += __shfl_xor_sync(0xffffffff, result2, k, warpSize);
  }

  /* element of r for which warp is responsible */
  i = threadIdx.y + blockIdx.y*blockDim.y;

  /* first thread in each warp sets the final result */
  if (i < m && threadIdx.x == 0) {
    r[i] = result1;
    if (i < n - 1) {
      r[i + m] = result2;
    }
  }
}

void convolve_v0(const int m, const int n, const float* p, const int incp,
    const float* q, const int incq, float* r, const int incr) {
  const float *p1 = p, *q1 = q;
  int incp1 = incp, incq1 = incq;
  int m1 = m, n1 = n;
  if (n > m) {
    /* swap to put largest vector on the left */
    p1 = q;
    q1 = p;
    incp1 = incq;
    incq1 = incp;
    m1 = n;
    n1 = m;
  }
  dim3 block(BLOCK_SIZE);
  dim3 grid((m1 + n1 - 1 + block.x - 1)/block.x);
  kernel_convolve_v0<<<grid,block>>>(m1, n1, p1, incp1, q1, incq1, r, incr);
}

void convolve_v1(const int m, const int n, const float* p, const int incp,
    const float* q, const int incq, float* r, const int incr) {
  const float *p1 = p, *q1 = q;
  int incp1 = incp, incq1 = incq;
  int m1 = m, n1 = n;
  if (n > m) {
    /* swap to put largest vector on the left */
    p1 = q;
    q1 = p;
    incp1 = incq;
    incq1 = incp;
    m1 = n;
    n1 = m;
  }
  dim3 block(BLOCK_SIZE);
  dim3 grid((m1 + block.x - 1)/block.x);
  kernel_convolve_v1<<<grid,block>>>(m1, n1, p1, incp1, q1, incq1, r, incr);
}

void convolve_v2(const int m, const int n, const float* p, const int incp,
    const float* q, const int incq, float* r, const int incr) {
  const float *p1 = p, *q1 = q;
  int incp1 = incp, incq1 = incq;
  int m1 = m, n1 = n;
  if (n > m) {
    /* swap to put largest vector on the left */
    p1 = q;
    q1 = p;
    incp1 = incq;
    incq1 = incp;
    m1 = n;
    n1 = m;
  }
  dim3 block(BLOCK_SIZE);
  dim3 grid((m1 + block.x - 1)/block.x);
  size_t shared = block.x*sizeof(float);
  kernel_convolve_v2<<<grid,block,shared>>>(m1, n1, p1, incp1, q1, incq1, r, incr);
}

void convolve_v3(const int m, const int n, const float* p, const int incp,
    const float* q, const int incq, float* r, const int incr) {
  const float *p1 = p, *q1 = q;
  int incp1 = incp, incq1 = incq;
  int m1 = m, n1 = n;
  if (n > m) {
    /* swap to put largest vector on the left */
    p1 = q;
    q1 = p;
    incp1 = incq;
    incq1 = incp;
    m1 = n;
    n1 = m;
  }
  dim3 block(BLOCK_SIZE);
  dim3 grid((m1 + block.x - 1)/block.x);
  size_t shared = 3*block.x*sizeof(float);
  kernel_convolve_v3<<<grid,block,shared>>>(m1, n1, p1, incp1, q1, incq1, r, incr);
}

void convolve_v4(const int m, const int n, const float* p, const int incp,
    const float* q, const int incq, float* r, const int incr) {
  const float *p1 = p, *q1 = q;
  int incp1 = incp, incq1 = incq;
  int m1 = m, n1 = n;
  if (n > m) {
    /* swap to put largest vector on the left */
    p1 = q;
    q1 = p;
    incp1 = incq;
    incq1 = incp;
    m1 = n;
    n1 = m;
  }
  dim3 block(32, 32);
  dim3 grid(1, (m1 + block.y - 1)/block.y);
  size_t shared = 3*block.x*block.y*sizeof(float);
  kernel_convolve_v4<<<grid,block,shared>>>(m1, n1, p1, incp1, q1, incq1, r, incr);
}

}
