#include "hip/hip_runtime.h"
#include <config.h>
#include <cassert>

/* nvcc compiles *.cu files as C++, whereas the remainder of our source code
 * is in C; this ensures that we get C external linkage so that we can easily
 * call the functions in this file from C code, not just C++ code */
extern "C" {

__global__ void kernel_convolve_v0(const int m, const int n, const float* p,
    const int incp, const float* q, const int incq, float* r,
    const int incr) {
  /* element of r for which thread is responsible */
  int i = threadIdx.y + blockIdx.y*blockDim.y;

  if (i < m + n - 1) {
    float result = 0.0f;
    for (int j = 0; j < n; ++j) {
      if (0 <= i - j && i - j < m) {
        result += p[(i - j)*incp]*q[j*incq];
      }
    }
    r[i] = result;
  }
}

__global__ void kernel_convolve_v1(const int m, const int n, const float* p,
    const int incp, const float* q, const int incq, float* r,
    const int incr) {
  /* element of r for which thread is responsible */
  int i = threadIdx.y + blockIdx.y*blockDim.y;
  
  if (i < m) {
    float result1 = 0.0f, result2 = 0.0f;
    for (int j = 0; j < n; ++j) {
      if (0 <= i - j) {
        result1 += p[(i - j)*incp]*q[j*incq];
      } else {
        result2 += p[(m + i - j)*incp]*q[j*incq];
      }
    }
    r[i] = result1;
    if (i < n - 1) {
      r[i + m] = result2;
    }
  }
}

__global__ void kernel_convolve_v2(const int m, const int n, const float* p,
    const int incp, const float* q, const int incq, float* r,
    const int incr) {
  /* shared memory */
  extern __shared__ float q_shared[];

  /* element of r for which thread is responsible */
  int i = threadIdx.y + blockIdx.y*blockDim.y;

  float result1 = 0.0f, result2 = 0.0f;
  for (int base = 0; base < n; base += blockDim.y) {
    int j = threadIdx.y;
    __syncthreads();
    q_shared[j] = (base + j < n) ? q[(base + j)*incq] : 0.0f;
    __syncthreads();

    if (i < m) {
      for (j = 0; j < blockDim.y; ++j) {
        if (0 <= i - base - j) {
          result1 += p[(i - base - j)*incp]*q_shared[j];
        } else {
          result2 += p[(m + i - base - j)*incp]*q_shared[j];
        }
      }
    }
  }
  if (i < m) {
    r[i] = result1;
    if (i < n - 1) {
      r[i + m] = result2;
    }
  }
}

__global__ void kernel_convolve_v3(const int m, const int n, const float* p,
    const int incp, const float* q, const int incq, float* r,
    const int incr) {
  assert(blockDim.x == warpSize && gridDim.x == 1);

  /* shared memory */
  extern __shared__ float q_shared[];

  /* element of r for which warp is responsible */
  int i = threadIdx.y + blockIdx.y*blockDim.y;

  float result1 = 0.0f, result2 = 0.0f;
  for (int base = 0; base < n; base += warpSize*blockDim.y) {
    int j = threadIdx.y*warpSize + threadIdx.x;
    __syncthreads();
    q_shared[j] = (base + j < n) ? q[(base + j)*incq] : 0.0f;
    __syncthreads();

    if (i < m) {
      for (j = threadIdx.x; j < warpSize*blockDim.y; j += warpSize) {
        if (0 <= i - base - j) {
          result1 += p[(i - base - j)*incp]*q_shared[j];
        } else {
          result2 += p[(m + i - base - j)*incp]*q_shared[j];
        }
      }
    }
  }

  /* sum across threads of warp, using butterfly sum */
  for (int k = 16; k >= 1; k /= 2) {
    result1 += __shfl_xor_sync(0xffffffff, result1, k, warpSize);
    result2 += __shfl_xor_sync(0xffffffff, result2, k, warpSize);
  }

  /* set the final result, only first thread in each warp */
  if (i < m && threadIdx.x == 0) {
    r[i] = result1;
    if (i < n - 1) {
      r[i + m] = result2;
    }
  }
}

__global__ void kernel_convolve_v4(const int m, const int n, const float* p,
    const int incp, const float* q, const int incq, float* r,
    const int incr) {
  /* shared memory */
  extern __shared__ float shared[];
  float* q_shared = shared;
  float* p_shared = q_shared + 2*blockDim.y;  // permits -ve indices

  /* element of r for which thread is responsible */
  int base_i = blockIdx.y*blockDim.y;
  int base_j = 0;
  int i = threadIdx.y;
  int j = threadIdx.y;
  int k;
  float result1 = 0.0f, result2 = 0.0f;
  while (base_j < n) {
    __syncthreads();
    k = base_j + j;
    q_shared[j] = (k < n) ? q[k*incq] : 0.0f;

    k = base_i + i;
    if (k < m) {
      if (k >= 0) {
        p_shared[i] = p[k*incp];
      } else {
        p_shared[i] = p[m + k*incp];
      }
    } else {
      p_shared[i] = 0.0f;
    }

    k = base_i + i - blockDim.y;
    if (k < m) {
      if (k >= 0) {
        p_shared[i - blockDim.y] = p[k*incp];
      } else {
        p_shared[i - blockDim.y] = p[m + k*incp];
      }
    } else {
      p_shared[i - blockDim.y] = 0.0f;
    }
    __syncthreads();

    for (k = 0; k < blockDim.y; ++k) {
      if (0 <= base_i + i - k) {
        result1 += p_shared[i - k]*q_shared[k];
      } else {
        result2 += p_shared[i - k]*q_shared[k];        
      }
    }

    base_i -= blockDim.y;
    base_j += blockDim.y;
  }

  i = threadIdx.y + blockIdx.y*blockDim.y;
  if (i < m) {
    r[i] = result1;
    if (i < n - 1) {
      r[i + m] = result2;
    }
  }
}

void convolve_v0(const int m, const int n, const float* p, const int incp,
    const float* q, const int incq, float* r, const int incr) {
  const float *p1 = p, *q1 = q;
  int incp1 = incp, incq1 = incq;
  int m1 = m, n1 = n;
  if (n > m) {
    /* swap to put largest vector on the left */
    p1 = q;
    q1 = p;
    incp1 = incq;
    incq1 = incp;
    m1 = n;
    n1 = m;
  }
  dim3 block(1, BLOCK_SIZE);
  dim3 grid(1, (m1 + n1 - 1 + block.y - 1)/block.y);
  kernel_convolve_v0<<<grid,block>>>(m1, n1, p1, incp1, q1, incq1, r, incr);
}

void convolve_v1(const int m, const int n, const float* p, const int incp,
    const float* q, const int incq, float* r, const int incr) {
  const float *p1 = p, *q1 = q;
  int incp1 = incp, incq1 = incq;
  int m1 = m, n1 = n;
  if (n > m) {
    /* swap to put largest vector on the left */
    p1 = q;
    q1 = p;
    incp1 = incq;
    incq1 = incp;
    m1 = n;
    n1 = m;
  }
  dim3 block(1, BLOCK_SIZE);
  dim3 grid(1, (m1 + block.y - 1)/block.y);
  kernel_convolve_v1<<<grid,block>>>(m1, n1, p1, incp1, q1, incq1, r, incr);
}

void convolve_v2(const int m, const int n, const float* p, const int incp,
    const float* q, const int incq, float* r, const int incr) {
  const float *p1 = p, *q1 = q;
  int incp1 = incp, incq1 = incq;
  int m1 = m, n1 = n;
  if (n > m) {
    /* swap to put largest vector on the left */
    p1 = q;
    q1 = p;
    incp1 = incq;
    incq1 = incp;
    m1 = n;
    n1 = m;
  }
  dim3 block(1, BLOCK_SIZE);
  dim3 grid(1, (m1 + block.y - 1)/block.y);
  size_t shared = block.y*sizeof(float);
  kernel_convolve_v2<<<grid,block,shared>>>(m1, n1, p1, incp1, q1, incq1, r, incr);
}

void convolve_v3(const int m, const int n, const float* p, const int incp,
    const float* q, const int incq, float* r, const int incr) {
  const float *p1 = p, *q1 = q;
  int incp1 = incp, incq1 = incq;
  int m1 = m, n1 = n;
  if (n > m) {
    /* swap to put largest vector on the left */
    p1 = q;
    q1 = p;
    incp1 = incq;
    incq1 = incp;
    m1 = n;
    n1 = m;
  }
  dim3 block(32, 16);
  dim3 grid(1, (m1 + block.y - 1)/block.y);
  size_t shared = block.x*block.y*sizeof(float);
  kernel_convolve_v3<<<grid,block,shared>>>(m1, n1, p1, incp1, q1, incq1, r, incr);
}

void convolve_v4(const int m, const int n, const float* p, const int incp,
    const float* q, const int incq, float* r, const int incr) {
  const float *p1 = p, *q1 = q;
  int incp1 = incp, incq1 = incq;
  int m1 = m, n1 = n;
  if (n > m) {
    /* swap to put largest vector on the left */
    p1 = q;
    q1 = p;
    incp1 = incq;
    incq1 = incp;
    m1 = n;
    n1 = m;
  }
  dim3 block(1, BLOCK_SIZE);
  dim3 grid(1, (m1 + block.y - 1)/block.y);
  size_t shared = 3*block.y*sizeof(float);
  kernel_convolve_v4<<<grid,block,shared>>>(m1, n1, p1, incp1, q1, incq1, r, incr);
}

}
