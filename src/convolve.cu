#include "hip/hip_runtime.h"
#include <config.h>
#include <cassert>

/* nvcc compiles *.cu files as C++, whereas the remainder of our source code
 * is in C; this ensures that we get C external linkage so that we can easily
 * call the functions in this file from C code, not just C++ code */
extern "C" {

__global__ void kernel_convolve_v0(const int m, const int n, const float* p,
    const int incp, const float* q, const int incq, float* r,
    const int incr) {
  /* element of r for which thread is responsible */
  int i = threadIdx.y + blockIdx.y*blockDim.y;
  
  /* sum across elements in each thread */
  float result = 0.0f;
  for (int j = 0; j < n; ++j) {
    if (0 <= i - j && i - j < m) {
      result += p[(i - j)*incp]*q[j*incq];
    }
  }
  
  /* write element */
  r[i] = result;
}

__global__ void kernel_convolve_v1(const int m, const int n, const float* p,
    const int incp, const float* q, const int incq, float* r,
    const int incr) {
  /* first element of r for which thread is responsible */
  int i = threadIdx.y + blockIdx.y*blockDim.y;
  
  /* sum across elements in each thread */
  float result1 = 0.0f, result2 = 0.0f;
  for (int j = 0; j < n; ++j) {
    if (0 <= i - j) {
      result1 += p[(i - j)*incp]*q[j*incq];
    } else {
      result2 += p[(m + i - j)*incp]*q[j*incq];
    }
  }
  
  /* write elements */
  r[i] = result1;
  if (i < m - 1) {
    r[i + m] = result2;
  }
}

__global__ void kernel_convolve_v2(const int m, const int n, const float* p,
    const int incp, const float* q, const int incq, float* r,
    const int incr) {
  __shared__ float* q_shared;
  int i = threadIdx.y + blockIdx.y*blockDim.y;
  float result1 = 0.0f, result2 = 0.0f;
  
  for (int base = 0; base < n; base += blockDim.y) {
    /* read the next batch into shared memory */
    int j = threadIdx.y;
    q_shared[j] = base + j < n ? q[(base + j)*incq] : 0.0f;
    __syncthreads();

    /* sum across elements in each thread  */
    for (j = 0; j < blockDim.y; ++j) {
      if (0 <= i - j) {
        result1 += p[(base + i - j)*incp]*q_shared[j];
      } else {
        result2 += p[(base + m + i - j)*incp]*q_shared[j];
      }
    }
  }

  /* set the final result */
  r[i] = result1;
  if (i < m - 1) {
    r[i + m] = result2;
  }
}

__global__ void kernel_convolve_v3(const int m, const int n, const float* p,
    const int incp, const float* q, const int incq, float* r,
    const int incr) {
  assert(blockDim.x == warpSize && gridDim.x == 1);

  __shared__ float* q_shared;
  int i = blockIdx.x*blockDim.x;
  float result1 = 0.0f, result2 = 0.0f;
  
  for (int base = 0; base < n; base += warpSize*blockDim.y) {
    /* read the next batch into shared memory */
    int j = threadIdx.y*warpSize + threadIdx.x;
    q_shared[j] = base + j < n ? q[(base + j)*incq] : 0.0f;
    __syncthreads();

    /* sum across elements in each thread  */
    for (j = threadIdx.x; j < warpSize*blockDim.y; j += warpSize) {
      /* read the next batch into shared memory */
      if (0 <= i - j) {
        result1 += p[(base + i - j)*incp]*q_shared[j];
      } else {
        result2 += p[(base + m + i - j)*incp]*q_shared[j];
      }
    }
  }

  /* sum across threads of warp, using butterfly sum */
  for (int k = 16; k >= 1; k /= 2) {
    result1 += __shfl_xor_sync(0xffffffff, result1, k, warpSize);
    result2 += __shfl_xor_sync(0xffffffff, result2, k, warpSize);
  }

  /* set the final result, only first thread in each warp */
  if (threadIdx.x == 0) {
    r[i] = result1;
    if (i < m - 1) {
      r[i + m] = result2;
    }
  }
}

void convolve_v0(const int m, const int n, const float* p, const int incp,
    const float* q, const int incq, float* r, const int incr) {
  dim3 block(1, BLOCK_SIZE);
  dim3 grid(1, (m + n - 1 + block.y - 1)/block.y);
  kernel_convolve_v0<<<grid,block>>>(m, n, p, incp, q, incq, r, incr);
}

void convolve_v1(const int m, const int n, const float* p, const int incp,
    const float* q, const int incq, float* r, const int incr) {
  dim3 block(1, BLOCK_SIZE);
  dim3 grid(1, (m + block.y - 1)/block.y);
  kernel_convolve_v1<<<grid,block>>>(m, n, p, incp, q, incq, r, incr);
}

void convolve_v2(const int m, const int n, const float* p, const int incp,
    const float* q, const int incq, float* r, const int incr) {
  dim3 block(1, BLOCK_SIZE);
  dim3 grid(1, (m + block.y - 1)/block.y);
  size_t shared = block.y*sizeof(float);
  kernel_convolve_v2<<<grid,block,shared>>>(m, n, p, incp, q, incq, r, incr);
}

void convolve_v3(const int m, const int n, const float* p, const int incp,
    const float* q, const int incq, float* r, const int incr) {
  dim3 block(32, BLOCK_SIZE);
  dim3 grid(1, (m + block.y - 1)/block.y);
  size_t shared = block.y*sizeof(float);
  kernel_convolve_v3<<<grid,block,shared>>>(m, n, p, incp, q, incq, r, incr);
}

}
