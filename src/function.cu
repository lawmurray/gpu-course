
#include <hip/hip_runtime.h>
static const float pi = 3.14159265358979f;

__global__ void kernel_rectify(int U, int B, float* Z, int ldZ) {
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  int j = blockIdx.y*blockDim.y + threadIdx.y;
  if (i < U && j < B) {
    Z[j*ldZ + i] = (Z[j*ldZ + i] <= 0.0f) ? 0.0f : Z[j*ldZ + i];
    // ^ ensures that NaN propagates rather than converts to zero
  }
}

__global__ void kernel_rectify_grad(int U, int B, const float* Z, int ldZ,
    float* dZ, int lddZ) {
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  int j = blockIdx.y*blockDim.y + threadIdx.y;
  if (i < U && j < B) {
    dZ[j*lddZ + i] = (Z[j*ldZ + i] <= 0.0f) ? 0.0f : dZ[j*lddZ + i];
  }
}

__global__ void kernel_log_likelihood(int B, const float* y, int incy,
    const float* Z, int ldZ, float* ll, int incll) {
  int j = blockIdx.y*blockDim.y + threadIdx.y;
  if (j < B) {
    float mu = Z[j*ldZ];
    float sigma = fabsf(Z[j*ldZ + 1]);
    float z = (y[j*incy] - mu)/sigma;
    ll[j*incll] = logf(2.0f/sqrtf(2.0f*pi)) - 0.5f*z*z - logf(sigma) -
        logf(erfcf(z/sqrtf(2.0f)));
  }
}

__global__ void kernel_log_likelihood_grad(int B, const float* y, int incy,
    const float* Z, int ldZ, float* dZ, int lddZ) {
  int j = blockIdx.y*blockDim.y + threadIdx.y;
  if (j < B) {
    float mu = Z[j*ldZ];
    float sigma = fabsf(Z[j*ldZ + 1]);
    float z = (y[j*incy] - mu)/sigma;
    float sqrt2 = sqrtf(2.0f);
    float tmp = (2.0f/sqrtf(pi))*expf(-0.5f*z*z)/erfcf(z/sqrt2);
    float dmu = z/sigma - tmp/(sigma*sqrt2);
    float dsigma = z/sigma - tmp*z/(sigma*sqrt2);

    dZ[j*lddZ] = dmu;
    dZ[j*lddZ + 1] = (Z[j*ldZ + 1] > 0.0f) ? dsigma : -dsigma;
  }
}

extern "C" void rectify(int U, int B, float* Z, int ldZ) {
  dim3 block(32, 16);
  dim3 grid((U + block.x - 1)/block.x, (B + block.y - 1)/block.y);
  kernel_rectify<<<grid,block>>>(U, B, Z, ldZ);
}

extern "C" void rectify_grad(int U, int B, const float* Z, int ldZ, float* dZ,
    int lddZ) {
  dim3 block(32, 16);
  dim3 grid((U + block.x - 1)/block.x, (B + block.y - 1)/block.y);
  kernel_rectify_grad<<<grid,block>>>(U, B, Z, ldZ, dZ, lddZ);
}

extern "C" void log_likelihood(int B, const float* y, int incy,
    const float* Z, int ldZ, float* l, int incl) {
  dim3 block(2, 256);
  dim3 grid(1, (B + block.y - 1)/block.y);
  kernel_log_likelihood<<<grid,block>>>(B, y, incy, Z, ldZ, l, incl);
}

extern "C" void log_likelihood_grad(int B, const float* y, int incy,
    const float* Z, int ldZ, float* dZ, int lddZ) {
  dim3 block(2, 256);
  dim3 grid(1, (B + block.y - 1)/block.y);
  kernel_log_likelihood<<<grid,block>>>(B, y, incy, Z, ldZ, dZ, lddZ);
}
